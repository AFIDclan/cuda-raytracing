#include "hip/hip_runtime.h"
﻿#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

#include "utils.hpp"
#include "Ray.hpp"
#include "TrianglePrimitive.hpp"
#include "MeshPrimitive.h"
#include "MeshInstance.hpp"

#include "OBJLoader.hpp"
#include <windows.h> // For SetCursorPos


__device__ Ray& raytrace(Ray& ray, MeshInstance* mesh_instances, int num_mesh_instances, d_MeshPrimitive* meshes)
{

    float hit_min = FLT_MAX;
    TrianglePrimitive hit_triangle;
    float3 hit_location;
    float3 hit_normal;

    for (int mesh_idx = 0; mesh_idx < num_mesh_instances; mesh_idx++) {

		MeshInstance mesh_instance = mesh_instances[mesh_idx];
        d_MeshPrimitive mesh = meshes[mesh_instance.mesh_index];

        // Express the ray direction in mesh coordinates
        float3 r_direction = apply_euler(mesh_instance.rotation, ray.direction);

        r_direction.x *= mesh_instance.inv_scale.x;
        r_direction.y *= mesh_instance.inv_scale.y;
        r_direction.z *= mesh_instance.inv_scale.z;

        // Express the ray origin in mesh coordinates
        float3 r_origin = apply_lre(mesh_instance.pose, ray.origin);

        // Eg. Scale of 2 --> multiply the origin times 0.5 and make the object appear 2x the size
        r_origin.x *= mesh_instance.inv_scale.x;
        r_origin.y *= mesh_instance.inv_scale.y;
        r_origin.z *= mesh_instance.inv_scale.z;

        Ray r_ray = Ray(
            r_origin,
            r_direction,
            ray.pixel
        );


        int stack[64];
        int stack_index = 0;

        // Start with the root node
        stack[stack_index++] = 0;

        while (stack_index > 0) {
            int node_index = stack[--stack_index];
            d_BVHTree current_bvh = mesh.bvh_top[node_index];

            // We are assuming this ray intersects with the bounding box of the node since it was pushed onto the stack

            if (current_bvh.child_index_a > 0) {
                // If the node has children, push them onto the stack

                float dist_a = mesh.bvh_top[current_bvh.child_index_a].ray_intersects(r_ray);
                float dist_b = mesh.bvh_top[current_bvh.child_index_b].ray_intersects(r_ray);

                if (dist_a < dist_b) {
                    if (dist_b < hit_min) stack[stack_index++] = current_bvh.child_index_b;
                    if (dist_a < hit_min) stack[stack_index++] = current_bvh.child_index_a;
                }
                else {
                    if (dist_a < hit_min) stack[stack_index++] = current_bvh.child_index_a;
                    if (dist_b < hit_min) stack[stack_index++] = current_bvh.child_index_b;
                }


            }
            else {
                // Leaf node: check for intersections with triangles
                for (int i = 0; i < current_bvh.count_triangles; i++) {
                    int index = current_bvh.triangle_indices[i];

                    float3 intersection = mesh.triangles[index].ray_intersect(r_ray);

                    // If the intersection is at FLT_MAX, the ray did not intersect with the triangle
                    if (intersection.x == FLT_MAX)
                        continue;

                    bool inside = mesh.triangles[index].point_inside(intersection);

                    if (inside) {
                        float distance = magnitude(intersection - r_ray.origin);

						// Positive means the ray is facing the same direction as the normal and we hit the back of the triangle
						float same_dir = dot(r_ray.direction, mesh.triangles[index].normal);

                        if (same_dir < 0 && (hit_min == -1.0f || distance < hit_min)) {
                            hit_min = distance;
                            hit_triangle = mesh.triangles[index];
                            

                            // Express normal in world coordinates
                            hit_normal = apply_euler(mesh_instance.inv_rotation, hit_triangle.normal);

                            hit_normal.x *= mesh_instance.scale.x;
                            hit_normal.y *= mesh_instance.scale.y;
                            hit_normal.z *= mesh_instance.scale.z;

							// Scaling the direction can un-normalize it
							hit_normal = normalize(hit_normal);

                            // Express the location in world coordinates
                            hit_location = apply_lre(mesh_instance.inv_pose, intersection);

                            hit_location.x *= mesh_instance.scale.x;
                            hit_location.y *= mesh_instance.scale.y;
                            hit_location.z *= mesh_instance.scale.z;
                        }
                    }
                }
            }
        }
    }

    if (hit_min != FLT_MAX) {

		
		// Move the ray to the hit location
        ray.origin = hit_location;

        // Reflect around normal
        ray.direction = ray.direction - (2.0f * (dot(ray.direction, hit_normal) * hit_normal));
		
		// Calculate the inverse of the direction
		ray.direction_inv.x = 1.0f / ray.direction.x;
		ray.direction_inv.y = 1.0f / ray.direction.y;
		ray.direction_inv.z = 1.0f / ray.direction.z;
   

        // Move just slightly so we don't capture the face we just hit
        ray.origin = ray.origin + ray.direction * 1e-4;

		// Calculate the loss of light due to the angle of incidence
        float cos_illum = dot(hit_normal, ray.direction);

		// Apply the color of the triangle to the ray
        ray.color.x *= (hit_triangle.color.x * cos_illum);
        ray.color.y *= (hit_triangle.color.y * cos_illum);
        ray.color.z *= (hit_triangle.color.z * cos_illum);


     } else {
        // Pale blue sky
        ray.color.x *= 1.0;
        ray.color.y *= 0.8;
        ray.color.z *= 0.6;

        ray.illumination = 1.0;

        ray.terminated = true;
    }

    return ray;
}


// Simple CUDA kernel to invert image colors
__global__ void render(uchar3* img, int width, int height, size_t pitch, const float3x3 K_inv, const lre camera_pose, MeshInstance* mesh_instances, int num_mesh_instances, d_MeshPrimitive* meshes) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
		return;
	}

    float3 origin = make_float3(camera_pose.x, camera_pose.y, camera_pose.z);

	float3 ph = make_float3(x, y, 1.0f);
	float3 direction = apply_matrix(K_inv, ph);


    // Rotate by 90 deg to make y forward (world space)
    direction = make_float3(direction.x, direction.z, -direction.y);

    // Apply the camera's pose to the direction
    direction = apply_euler(make_float3(camera_pose.yaw, camera_pose.pitch, camera_pose.roll), direction);
     
    //Camera Ray direction in world space
    direction = normalize(direction);

    Ray ray = Ray(
        origin,
        direction,
        make_uint2(x, y)
    );

    for (int i = 0; i < 4; i++)
    {
        if (ray.terminated)
            break;

        ray = raytrace(ray, mesh_instances, num_mesh_instances, meshes);
    }

    
    //ray = raytrace(ray, meshes, num_meshes);

    //if (!ray.terminated)
    //{
    //    ray.illumination = dot(ray.direction, direction);
    //}
   

    uchar3* row = (uchar3*)((char*)img + y * pitch);
    row[x].x = (ray.color.x * ray.illumination * 255);
    row[x].y = (ray.color.y * ray.illumination * 255);
    row[x].z = (ray.color.z * ray.illumination * 255);
    
	
}

void display_image(uchar3* d_img, int width, int height, size_t pitch, double fps)
{
    // Wrap the CUDA memory in an OpenCV GpuMat
    cv::cuda::GpuMat img_gpu(height, width, CV_8UC3, d_img, pitch);

    // Download the processed image back to host memory
    cv::Mat img_cpu;
    img_gpu.download(img_cpu);

    // Convert FPS to string and overlay it on the image
    std::string fps_text = "FPS: " + std::to_string(fps);
    cv::putText(img_cpu, fps_text, cv::Point(10, 30), cv::FONT_HERSHEY_SIMPLEX, 1.0, cv::Scalar(0, 255, 0), 2);

    // Display the image using OpenCV
    cv::imshow("Image", img_cpu);

    // Capture key pressed
    int key = cv::waitKey(1);

    // If the key pressed is 'q', then exit the loop
    if (key == 'q') {
        exit(0);
    }
}

struct MouseParams
{
    int last_x;
    int last_y;
    bool has_last = false;

    bool is_down = false;

    lre *pose;
};

void on_mouse(int event, int x, int y, int, void* param)
{
    // Cast the param back to the correct type
    MouseParams* mouse_state = static_cast<MouseParams*>(param);
    if (event == cv::EVENT_LBUTTONDOWN)
    {
        mouse_state->is_down = true;
    } else if (event == cv::EVENT_LBUTTONUP)
    {
        mouse_state->is_down = false;
    } else if (event == cv::EVENT_MOUSEMOVE)
    {
        
        if (mouse_state->has_last && mouse_state->is_down)
        {
            int dx = x - mouse_state->last_x;
            int dy = y - mouse_state->last_y;

            mouse_state->pose->yaw += dx * -0.001;
            mouse_state->pose->pitch += dy * 0.001;
        }

        mouse_state->last_x = x;
        mouse_state->last_y = y;
        mouse_state->has_last = true;
    }
}

int main() {

	//transforms::test_all();

	//exit(0);

    // Image dimensions

    double fps = 0.0;

    int64 start_time = 0;
    int64 end_time = 0;


    int width = 1280;
    int height = 720;

    float3x3 K = {
        800.0, 0.0, width / 2,
        0,     800.0, height / 2,
        0,     0,   1
    };

	float3x3 K_inv = invert_intrinsic(K);

    lre camera_pose = lre();


    camera_pose.x = 0;
    camera_pose.y = -8;
    camera_pose.z = 0;





    MeshPrimitive cow = OBJLoader::load("./cow.obj");
    MeshPrimitive teapot = OBJLoader::load("./teapot.obj");
    MeshPrimitive cube = OBJLoader::load("./cube.obj");

    teapot.bvh_top.print_stats();

    d_MeshPrimitive* d_meshes;

    hipMalloc(&d_meshes, sizeof(d_MeshPrimitive) * 3);

    hipMemcpy(&d_meshes[0], cow.to_device(), sizeof(d_MeshPrimitive), hipMemcpyHostToDevice);
    hipMemcpy(&d_meshes[1], teapot.to_device(), sizeof(d_MeshPrimitive), hipMemcpyHostToDevice);
    hipMemcpy(&d_meshes[2], cube.to_device(), sizeof(d_MeshPrimitive), hipMemcpyHostToDevice);

	MeshInstance cow_instance = MeshInstance(0);

    cow_instance.pose.x = -2;
	cow_instance.pose.pitch = 3.141592 / 2;
    cow_instance.scale = make_float3(0.2, 0.2, 0.2);


    MeshInstance cube_instance = MeshInstance(2);

    cube_instance.pose.z = -2;
    cube_instance.scale = make_float3(10.0, 10.0, 1.0);

	MeshInstance teapot_instance = MeshInstance(1);

	teapot_instance.pose.x = 2;
	teapot_instance.pose.pitch = 3.141592 / 2;

	

    cow_instance.build_inv();
    cube_instance.build_inv();
	teapot_instance.build_inv();


	MeshInstance* d_mesh_instances;

	hipMalloc(&d_mesh_instances, sizeof(MeshInstance) * 3);

	hipMemcpy(&d_mesh_instances[0], &cow_instance, sizeof(MeshInstance), hipMemcpyHostToDevice);
	hipMemcpy(&d_mesh_instances[1], &cube_instance, sizeof(MeshInstance), hipMemcpyHostToDevice);
	hipMemcpy(&d_mesh_instances[2], &teapot_instance, sizeof(MeshInstance), hipMemcpyHostToDevice);




    //MeshPrimitive teapot = OBJLoader::load("C:/workspace/CudaRaytracer/cube.obj");

	//teapot.set_world_position(make_float3(0, 0, 8));
	//teapot.set_world_position(make_float3(0, 8, -2));

	//teapot.set_world_rotation(make_float3(0, 0, 0));

   




    // Allocate CUDA memory for the image
    uchar3* d_img;
    size_t pitch;
    hipMallocPitch(&d_img, &pitch, width * sizeof(uchar3), height);

    // Define CUDA kernel launch configuration
    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

	float angle = 0.0f;

    MouseParams mouse_state;
    mouse_state.pose = &camera_pose;
    cv::namedWindow("Image");
    cv::setMouseCallback("Image", on_mouse, &mouse_state);

    // Loop while program is running
    while (true) {

		angle += 0.005f;

        // Start measuring time
        start_time = cv::getTickCount();

        teapot_instance.pose.yaw = angle;
		teapot_instance.build_inv();

        hipMemcpy(&d_mesh_instances[2], &teapot_instance, sizeof(MeshInstance), hipMemcpyHostToDevice);

        // Launch the CUDA kernel to invert colors
        render << <grid_size, block_size >> > (d_img, width, height, pitch, K_inv, camera_pose, d_mesh_instances, 3, d_meshes);
        hipDeviceSynchronize();

        // End measuring time
        end_time = cv::getTickCount();
        double time_taken = (end_time - start_time) / cv::getTickFrequency();
        fps = 1.0 / time_taken;

        display_image(d_img, width, height, pitch, fps);
    }

    // Free CUDA memory
    hipFree(d_img);

    return 0;
}
